#include "hip/hip_runtime.h"
#include <cassert>
#include <numeric>

#include <stdio.h>

#include "CudaFunctions.h"
#include "hip/hip_runtime.h"
#include ""


#include "PatternsLibrary.h"
#include "TracesExtraction.h"


#define CUDA_CHECK_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            return; \
        } \
    } while (0)

/*
__device__ void ProcessPixelDevice(uint16_t* pxls, uint32_t width, uint32_t i, uint32_t j, uint32_t* pxlK, float* pxlL, PatternsLibrary& lib)
{
    uint32_t pixelIdx = i * width + j;

    int   k_ast_max = -1;
    float L_ast_max = 0.f;
    const uint8_t* pattern = nullptr;
    
    for (int k = 0; k < PATTERNS_COUNT; ++k) {
        int WL;
        int kGroup;
        if (k < UP5_OFFSET) {
            WL = UP3_SIZE;
            kGroup = k - UP3_OFFSET;
            pattern = &(lib.up3[kGroup][0]);
        }
        else if (k < UP7_OFFSET) {
            WL = UP5_SIZE;
            kGroup = k - UP5_OFFSET;
            pattern = &(lib.up5[kGroup][0]);
        }
        else if (k < UP9_OFFSET) {
            WL = UP7_SIZE;
            kGroup = k - UP7_OFFSET;
            pattern = &(lib.up7[kGroup][0]);
        }
        else if (k < UP11_OFFSET) {
            WL = UP9_SIZE;
            kGroup = k - UP9_OFFSET;
            pattern = &(lib.up9[kGroup][0]);
        }
        else {
            WL = UP11_SIZE;
            kGroup = k - UP11_OFFSET;
            pattern = &(lib.up11[kGroup][0]);
        }
    

        float SY = 0.f;
        float SYF = 0.f;
    
        for (int p = -WL / 2; p <= WL / 2; ++p) {
            for (int q = -WL / 2; q <= WL / 2; ++q) {
                uint16_t pxlColor = pxls[(i + p) * width + (j + q)];
                SY += pxlColor;
                SYF += pxlColor * pattern[(p + WL / 2) * WL + (q + WL / 2)];
            }
        }
    
        float a_ast;
        float b_ast;
        float L_ast;
    
        const int S = WL * WL;
    
        if (kGroup == 0) {
            a_ast = 0;
            b_ast = SY / S;
            L_ast = SY * SY / S;
        }
        else {
            a_ast = lib.C_a[k] * (SYF * S - SY * lib.SF[k]);
            b_ast = lib.C_b[k] * (SY - a_ast * lib.SF[k]);
            L_ast = 2 * a_ast * SYF - a_ast * a_ast * lib.SFF[k] + b_ast * b_ast * S;
        }
    
        if (a_ast > 0.f && L_ast_max < L_ast || k_ast_max == -1) {
            L_ast_max = L_ast;
            k_ast_max = k;
        }
    }
    
    
    *pxlK = k_ast_max;
    *pxlL = L_ast_max;
}

__device__ void ProcessPixelGPU(void* argsPtr, void* retPtr, unsigned int i, unsigned int j) {
    int pixelIdx = i * 32 + j;
    CudaOutput* pOutput = static_cast<CudaOutput*>(retPtr);

    if (pOutput != nullptr && pOutput->K != nullptr && pOutput->L != nullptr) {
        pOutput->K[0] = 3;
        int temp = pOutput->K[0] * 3;
        float denominator = static_cast<float>(temp);
        pOutput->L[0] = 0.f / denominator;
    }

    //int   k_ast_max = -1;
    //float L_ast_max = 0.f;
    //
    ////printf("Processing pixel (%u, %u)\n", i, j);
    ////PatternsLibrary* pLib = &(pInput->lib);
    ////const uint8_t* pattern = &(pLib->up3[0][0]);
    //
    //pOutput->K[0] = 3;
    //int kGroup = 0;

    //for (int k = 0; k < 1; ++k) {
    //    int WL;
    //    int kGroup;
    //    if (k < UP5_OFFSET) {
    //        WL = UP3_SIZE;
    //        kGroup = k - UP3_OFFSET;
    //        pattern = &(pLib->up3[kGroup][0]);
    //    }
    //    if (k >= UP5_OFFSET && k < UP7_OFFSET) {
    //        WL = UP5_SIZE;
    //        kGroup = k - UP5_OFFSET;
    //        pattern = &(pLib->up5[kGroup][0]);
    //    }
    //    if (k >= UP7_OFFSET && k < UP9_OFFSET) {
    //        WL = UP7_SIZE;
    //        //kGroup = k - UP7_OFFSET;
    //        //pattern = &(pLib->up7[kGroup][0]);
    //    }
    //    else if (k < UP11_OFFSET) {
    //        WL = UP9_SIZE;
    //        kGroup = k - UP9_OFFSET;
    //        pattern = &(pLib->up9[kGroup][0]);
    //    }
    //    else {
    //        WL = UP11_SIZE;
    //        kGroup = k - UP11_OFFSET;
    //        pattern = &(pLib->up11[kGroup][0]);
    //    }
    //
    //    float SY = 0.f;
    //    float SYF = 0.f;
    //    ////
    //    ////for (int p = -WL / 2; p <= WL / 2; p = p + 1) {
    //    ////    for (int q = -WL / 2; q <= WL / 2; q = q + 1) {
    //    ////        //uint16_t pxlColor = pInput->pxls[(i + p) * pInput->width + (j + q)];
    //    ////        //SY += pxlColor;
    //    ////        //SYF += pxlColor * pattern[(p + WL / 2) * WL + (q + WL / 2)];
    //    ////    }
    //    ////}
    //    ////
    //    float a_ast;
    //    float b_ast;
    //    float L_ast;
    //    //
    //    int S = pOutput->K[0] * pOutput->K[0];
    //    //
    //    if (true) {
    //        //a_ast = 0;
    //        
    //        //b_ast = SY / S;
    //        //L_ast = SY * SY / S;
    //    }
    //    //else {
    //    //    a_ast = pLib->C_a[k] * (SYF * S - SY * pLib->SF[k]);
    //    //    b_ast = pLib->C_b[k] * (SY - a_ast * pLib->SF[k]);
    //    //    L_ast = 2 * a_ast * SYF - a_ast * a_ast * pLib->SFF[k] + b_ast * b_ast * S;
    //    //}
    //    //
    //    //if (a_ast > 0.f && L_ast_max < L_ast || k_ast_max == -1) {
    //    //    L_ast_max = L_ast;
    //    //    k_ast_max = k;
    //    //}
    ////}

    //ProcessPixelDevice(pInput->pxls, 1024, blockIdx, threadIdx, pOutput->K, pOutput->L, &(pInput->lib));
}
__device__ void (*deviceFuncPtr)(void*, void*, unsigned int, unsigned int) = ProcessPixelGPU;
*/


__global__ void ProcessPixelGPU(PixelType* pxls, uint32_t width, uint32_t pxlFrom, uint32_t pxlTo, uint32_t* pxlK, float* pxlL, PatternsLibrary* pLib)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + pxlFrom;
    if (idx >= pxlTo) return;

    uint32_t widthToProcess = (width - PATTERN_MAX_SIZE + (PATTERN_MAX_SIZE & 0x01u));
    uint16_t i = PATTERN_MAX_SIZE / 2 + idx / widthToProcess;
    uint16_t j = PATTERN_MAX_SIZE / 2 + idx % widthToProcess;

    int   k_ast_max = -1;
    float L_ast_max = 0.f;
    const uint8_t* pattern = nullptr;

    for (int k = 0; k < PATTERNS_COUNT; ++k) {
        int WL;
        int kGroup;
        if (k < UP5_OFFSET) {
            WL = UP3_SIZE;
            kGroup = k - UP3_OFFSET;
            pattern = &(pLib->up3[kGroup][0]);
        }
        else if (k < UP7_OFFSET) {
            WL = UP5_SIZE;
            kGroup = k - UP5_OFFSET;
            pattern = &(pLib->up5[kGroup][0]);
        }
        else if (k < UP9_OFFSET) {
            WL = UP7_SIZE;
            kGroup = k - UP7_OFFSET;
            pattern = &(pLib->up7[kGroup][0]);
        }
        else if (k < UP11_OFFSET) {
            WL = UP9_SIZE;
            kGroup = k - UP9_OFFSET;
            pattern = &(pLib->up9[kGroup][0]);
        }
        else {
            WL = UP11_SIZE;
            kGroup = k - UP11_OFFSET;
            pattern = &(pLib->up11[kGroup][0]);
        }

        float SY = 0.f;
        float SYF = 0.f;

        for (int p = -WL / 2; p <= WL / 2; ++p) {
            for (int q = -WL / 2; q <= WL / 2; ++q) {
                PixelType pxlColor = pxls[(i + p) * width + (j + q)];
                SY += pxlColor;
                SYF += pxlColor * pattern[(p + WL / 2) * WL + (q + WL / 2)];
            }
        }

        float a_ast;
        float b_ast;
        float L_ast;

        const int S = WL * WL;

        if (kGroup == 0) {
            a_ast = 0;
            b_ast = SY / S;
            L_ast = SY * SY / S;
        }
        else {
            a_ast = pLib->C_a[k] * (SYF * S - SY * pLib->SF[k]);
            b_ast = pLib->C_b[k] * (SY - a_ast * pLib->SF[k]);
            L_ast = 2 * a_ast * SYF - a_ast * a_ast * pLib->SFF[k] + b_ast * b_ast * S;
        }

        if (a_ast > 0.f && L_ast_max < L_ast || k_ast_max == -1) {
            L_ast_max = L_ast;
            k_ast_max = k;
        }
    }


    *(pxlK + (idx - pxlFrom)) = k_ast_max;
    *(pxlL + (idx - pxlFrom)) = L_ast_max;
}


extern "C" void ProcessImageCuda(uint16_t* pixels, uint32_t height, uint32_t width, uint32_t pxlFrom, uint32_t pxlTo, uint32_t * K, float* L, uint32_t KLsize, PatternsLibrary* pLib) {
    const int THREADS_BLOCK_SIZE = 256;
    const int GRID_SIZE = (pxlTo - pxlFrom) / THREADS_BLOCK_SIZE + 1;

    CUDA_CHECK_ERROR(hipSetDevice(0));

    uint16_t*        devPixels = nullptr;
    uint32_t*        devK      = nullptr;
    float*           devL      = nullptr;
    PatternsLibrary* devLib    = nullptr;

    CUDA_CHECK_ERROR(hipMalloc((void**)&devPixels, height * width * sizeof(uint16_t)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&devK, (pxlTo - pxlFrom) * sizeof(uint32_t)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&devL, (pxlTo - pxlFrom) * sizeof(float)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&devLib, sizeof(PatternsLibrary)));
    CUDA_CHECK_ERROR(hipMemcpy(devPixels, pixels, height * width * sizeof(uint16_t), hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(devLib,    pLib,   sizeof(PatternsLibrary),           hipMemcpyHostToDevice));

    ProcessPixelGPU <<<GRID_SIZE, THREADS_BLOCK_SIZE>>> (devPixels, width, pxlFrom, pxlTo, devK, devL, devLib);

    // Check for any errors launching the kernel
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    
    CUDA_CHECK_ERROR(hipMemcpy(K, devK, (pxlTo - pxlFrom) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipMemcpy(L, devL, (pxlTo - pxlFrom) * sizeof(float),    hipMemcpyDeviceToHost));

    hipFree(devPixels);
    hipFree(devK);
    hipFree(devL);
    hipFree(devLib);
}


/*
CudaFuncPtr GetDeviceFuncPtr() {
	CudaFuncPtr hostFuncPtr;
	hipError_t res = hipMemcpyFromSymbol(&hostFuncPtr, HIP_SYMBOL(deviceFuncPtr), sizeof(hostFuncPtr));
	assert(res == hipSuccess);
	return hostFuncPtr;
}
*/



/*
void runFunction(CudaFuncPtr func) {
	CudaFuncPtr pFunc = GetDeviceFuncPtr();
	runKernel << <1, 1 >> > (func);

	hipError_t res = hipDeviceSynchronize();
	assert(res == hipSuccess);
}
*/
