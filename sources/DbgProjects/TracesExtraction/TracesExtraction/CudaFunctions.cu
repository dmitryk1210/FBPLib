#include "hip/hip_runtime.h"
#include <cassert>
#include <numeric>

#include <stdio.h>

#include "CudaFunctions.h"
#include "hip/hip_runtime.h"
#include ""


#include "PatternsLibrary.h"
#include "TracesExtraction.h"


#define CUDA_CHECK_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            return; \
        } \
    } while (0)


__global__ void ProcessPixelGPU(PixelType* pxls, uint32_t width, uint32_t pxlFrom, uint32_t pxlTo, uint32_t* pxlK, float* pxlL, PatternsLibrary* pLib)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + pxlFrom;
    if (idx >= pxlTo) return;

    uint32_t widthToProcess = (width - PATTERN_MAX_SIZE + (PATTERN_MAX_SIZE & 0x01u));
    uint16_t i = PATTERN_MAX_SIZE / 2 + idx / widthToProcess;
    uint16_t j = PATTERN_MAX_SIZE / 2 + idx % widthToProcess;

    int   k_ast_max = -1;
    float L_ast_max = 0.f;
    const uint8_t* pattern = nullptr;

    for (int k = 0; k < PATTERNS_COUNT; ++k) {
        int WL;
        int kGroup;
        if (k < UP5_OFFSET) {
            WL = UP3_SIZE;
            kGroup = k - UP3_OFFSET;
            pattern = &(pLib->up3[kGroup][0]);
        }
        else if (k < UP7_OFFSET) {
            WL = UP5_SIZE;
            kGroup = k - UP5_OFFSET;
            pattern = &(pLib->up5[kGroup][0]);
        }
        else if (k < UP9_OFFSET) {
            WL = UP7_SIZE;
            kGroup = k - UP7_OFFSET;
            pattern = &(pLib->up7[kGroup][0]);
        }
        else if (k < UP11_OFFSET) {
            WL = UP9_SIZE;
            kGroup = k - UP9_OFFSET;
            pattern = &(pLib->up9[kGroup][0]);
        }
        else {
            WL = UP11_SIZE;
            kGroup = k - UP11_OFFSET;
            pattern = &(pLib->up11[kGroup][0]);
        }

        float SY = 0.f;
        float SYF = 0.f;

        for (int p = -WL / 2; p <= WL / 2; ++p) {
            for (int q = -WL / 2; q <= WL / 2; ++q) {
                PixelType pxlColor = pxls[(i + p) * width + (j + q)];
                SY += pxlColor;
                SYF += pxlColor * pattern[(p + WL / 2) * WL + (q + WL / 2)];
            }
        }

        float a_ast;
        float b_ast;
        float L_ast;

        const int S = WL * WL;

        if (kGroup == 0) {
            a_ast = 0;
            b_ast = SY / S;
            L_ast = SY * SY / S;
        }
        else {
            a_ast = pLib->C_a[k] * (SYF * S - SY * pLib->SF[k]);
            b_ast = pLib->C_b[k] * (SY - a_ast * pLib->SF[k]);
            L_ast = 2 * a_ast * SYF - a_ast * a_ast * pLib->SFF[k] + b_ast * b_ast * S;
        }

        if (a_ast > 0.f && L_ast_max < L_ast || k_ast_max == -1) {
            L_ast_max = L_ast;
            k_ast_max = k;
        }
    }


    *(pxlK + (idx - pxlFrom)) = k_ast_max;
    *(pxlL + (idx - pxlFrom)) = L_ast_max;
}


extern "C" void ProcessImageCuda(uint16_t* pixels, uint32_t height, uint32_t width, uint32_t pxlFrom, uint32_t pxlTo, uint32_t * K, float* L, uint32_t KLsize, PatternsLibrary* pLib) {
    const int THREADS_BLOCK_SIZE = 256;
    const int GRID_SIZE = (pxlTo - pxlFrom) / THREADS_BLOCK_SIZE + 1;

    CUDA_CHECK_ERROR(hipSetDevice(0));

    uint16_t*        devPixels = nullptr;
    uint32_t*        devK      = nullptr;
    float*           devL      = nullptr;
    PatternsLibrary* devLib    = nullptr;

    CUDA_CHECK_ERROR(hipMalloc((void**)&devPixels, height * width * sizeof(uint16_t)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&devK, (pxlTo - pxlFrom) * sizeof(uint32_t)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&devL, (pxlTo - pxlFrom) * sizeof(float)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&devLib, sizeof(PatternsLibrary)));
    CUDA_CHECK_ERROR(hipMemcpy(devPixels, pixels, height * width * sizeof(uint16_t), hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(devLib,    pLib,   sizeof(PatternsLibrary),           hipMemcpyHostToDevice));

    ProcessPixelGPU <<<GRID_SIZE, THREADS_BLOCK_SIZE>>> (devPixels, width, pxlFrom, pxlTo, devK, devL, devLib);

    // Check for any errors launching the kernel
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    
    CUDA_CHECK_ERROR(hipMemcpy(K, devK, (pxlTo - pxlFrom) * sizeof(uint32_t), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipMemcpy(L, devL, (pxlTo - pxlFrom) * sizeof(float),    hipMemcpyDeviceToHost));

    hipFree(devPixels);
    hipFree(devK);
    hipFree(devL);
    hipFree(devLib);
}
