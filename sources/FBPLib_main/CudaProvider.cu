#include <cassert>
#include <stdio.h>

#include "CudaProvider.h"
#include "hip/hip_runtime.h"


extern "C" bool HasCudaDevice() {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, 0);
    return err == hipSuccess;
}
