// CudaProvider.cu
//

#include "fbpGlobalDefines.h"

#ifdef USE_CUDA
#include <cassert>
#include <stdio.h>

#include "CudaProvider.h"
#include "hip/hip_runtime.h"


extern "C" bool HasCudaDevice() {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, 0);
    return err == hipSuccess;
}
#endif // USE_CUDA
